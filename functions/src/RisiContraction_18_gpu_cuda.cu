#include "hip/hip_runtime.h"
// Framework: GraphFlow
// Class: RisiContraction_18_gpu
// Author: Machine Learning Group of UChicago
// Institution: Department of Computer Science, The University of Chicago
// Copyright 2017 (c) UChicago. All rights reserved.

#include <THC.h>
#include <THCGeneral.h>
#include <hiprand.h>
#include <stdio.h>
#include <math.h>
#include <float.h>

#include "RisiContraction_18_gpu_cuda.h"

#ifdef __cplusplus
	extern "C" {
#endif

#define NUM_THREADS 512
#define BLOCK 512
#define NUM_CONTRACTIONS 18

__global__ void RisiContraction_18_forward_job(float *tensor, float *adj, float *value, int N, int nChanels) {
	
	__shared__ int nContractions;
	__shared__ int A;
	__shared__ int B;
	__shared__ int C;
	__shared__ int Y;


	nContractions = NUM_CONTRACTIONS;

	int global_threadId = blockIdx.x * blockDim.x + threadIdx.x;
	
	if (global_threadId < N * N * nChanels * nContractions) {	
		C = nChanels;
		B = N * C;
		A = N * B;

		Y = nChanels * nContractions;
		
		int f = (global_threadId % Y) % nChanels;
		int Case = (global_threadId % Y) / nChanels + 1;
		int y = (global_threadId / Y) % N;
		int x = (global_threadId / Y) / N;

		int a, b, c, d, e;
		float adj_value;

		float sum = 0.0;

		// +-----------+
		// | 1 + 1 + 1 |
		// +-----------+
		
		// Case 1 (1/50): Fix a, b. Contract c, d, e.
		if (Case == 1) {
			a = x;
			b = y;

			for (d = 0; d < N; ++d) {
				for (e = 0; e < N; ++e) {
					adj_value = adj[d * N + e];
					if (adj_value > 0) {
						for (c = 0; c < N; ++c) {
							sum += tensor[a * A + b * B + c * C + f] * adj_value;
						}
					}
				}
			}
		}

				
		// Case 2 (3/50): Fix a, d. Contract b, c, e.
		if (Case == 2) {		
			a = x;
			d = y;

			for (e = 0; e < N; ++e) {
				adj_value = adj[d * N + e];
				if (adj_value > 0) {
					for (b = 0; b < N; ++b) {
						for (c = 0; c < N; ++c) {
							sum += tensor[a * A + b * B + c * C + f] * adj_value;
						}
					}
				}
			}	
		}
		
		// Case 3 (5/50): Fix b, c. Contract a, d, e.
		if (Case == 3) {		
			b = x;
			c = y;

			for (d = 0; d < N; ++d) {
				for (e = 0; e < N; ++e) {
					adj_value = adj[d * N + e];
					if (adj_value > 0) {
						for (a = 0; a < N; ++a) {
							sum += tensor[a * A + b * B + c * C + f] * adj_value;
						}
					}
				}
			}	
		}

		// Case 4 (6/50): Fix b, d. Contract a, c, e.
		if (Case == 4) {
			b = x;
			d = y;

			for (e = 0; e < N; ++e) {
				adj_value = adj[d * N + e];
				if (adj_value > 0) {
					for (a = 0; a < N; ++a) {
						for (c = 0; c < N; ++c) {
							sum += tensor[a * A + b * B + c * C + f] * adj_value;
						}
					}
				}
			}
		}

		// Case 5 (10/50): Fix d, e. Contract a, b, c.
		if (Case == 5) {		
			d = x;
			e = y;

			adj_value = adj[d * N + e];
			if (adj_value > 0) {
				for (a = 0; a < N; ++a) {
					for (b = 0; b < N; ++b) {
						for (c = 0; c < N; ++c) {
							sum += tensor[a * A + b * B + c * C + f] * adj_value;
						}
					}
				}
			}
		}

		// +-------+
		// | 1 + 2 |
		// +-------+

		// Case 6 (11/50): (a, b). Contract (c, d). Singleton (e).
		if (Case == 6) {
			a = x;
			b = y;

			for (d = 0; d < N; ++d) {
				for (e = 0; e < N; ++e) {
					adj_value = adj[d * N + e];
					c = d;
					sum += tensor[a * A + b * B + c * C + f] * adj_value;
				}
			}
		}

		// Case 7 (13/50): (a, b). Contract (d, e). Singleton (c).
		if (Case == 7) {
			a = x;
			b = y;

			for (d = 0; d < N; ++d) {
				e = d;
				adj_value = adj[d * N + e];
				if (adj_value > 0) {
					for (c = 0; c < N; ++c) {
						sum += tensor[a * A + b * B + c * C + f] * adj_value;
					}
				}
			}
		}

		// Case 8 (17/50): (a, d). Contract (b, c). Singleton (e).
		if (Case == 8) {
			a = x;
			d = y;

			for (e = 0; e < N; ++e) {
				adj_value = adj[d * N + e];
				if (adj_value > 0) {
					for (b = 0; b < N; ++b) {
						c = b;
						sum += tensor[a * A + b * B + c * C + f] * adj_value;
					}
				}
			}
		}

		// Case 9 (18/50): (a, d). Contract (b, e). Singleton (c).
		if (Case == 9) {
			a = x;
			d = y;

			for (e = 0; e < N; ++e) {
				adj_value = adj[d * N + e];
				if (adj_value > 0) {
					b = e;
					for (c = 0; c < N; ++c) {
						sum += tensor[a * A + b * B + c * C + f] * adj_value;
					}
				}
			}
		}

		// Case 10 (23/50): (b, c). Contract (a, d). Singleton (e).
		if (Case == 10) {
			b = x;
			c = y;

			for (d = 0; d < N; ++d) {
				for (e = 0; e < N; ++e) {
					adj_value = adj[d * N + e];
					if (adj_value > 0) {
						a = d;
						sum += tensor[a * A + b * B + c * C + f] * adj_value;
					}
				}
			}
		}

		// Case 11 (26/50): (b, d). Contract (a, c). Singleton (e).
		if (Case == 11) {
			b = x;
			d = y;

			for (e = 0; e < N; ++e) {
				adj_value = adj[d * N + e];
				if (adj_value > 0) {
					for (a = 0; a < N; ++a) {
						c = a;
						sum += tensor[a * A + b * B + c * C + f] * adj_value;
					}
				}
			}
		}

		// Case 12 (27/50): (b, d). Contract (a, e). Singleton (c).
		if (Case == 12) {
			b = x;
			d = y;

			for (e = 0; e < N; ++e) {
				adj_value = adj[d * N + e];
				if (adj_value > 0) {
					a = e;
					for (int c = 0; c < N; ++c) {
						sum += tensor[a * A + b * B + c * C + f] * adj_value;
					}
				}
			}
		}

		// Case 13 (28/50): (b, d). Contract (c, e). Singleton (a).
		if (Case == 13) {
			b = x;
			d = y;

			for (e = 0; e < N; ++e) {
				adj_value = adj[d * N + e];
				if (adj_value > 0) {
					c = e;
					for (int a = 0; a < N; ++a) {
						sum += tensor[a * A + b * B + c * C + f] * adj_value;
					}
				}
			}
		}

		// Case 14 (38/50): (d, e). Contract (a, b). Singleton (c).
		if (Case == 14) {
			d = x;
			e = y;

			adj_value = adj[d * N + e];
			if (adj_value > 0) {
				for (int a = 0; a < N; ++a) {
					b = a;
					for (int c = 0; c < N; ++c) {
						sum += tensor[a * A + b * B + c * C + f] * adj_value;
					}
				}
			}
		}

		// Case 15 (40/50): (d, e). Contract (b, c). Singleton (a).
		if (Case == 15) {
			d = x;
			e = y;

			adj_value = adj[d * N + e];
			if (adj_value > 0) {
				for (int b = 0; b < N; ++b) {
					c = b;
					for (int a = 0; a < N; ++a) {
						sum += tensor[a * A + b * B + c * C + f] * adj_value;
					}
				}
			}
		}

		// +---+
		// | 3 |
		// +---+

		// Case 16 (43/50): (a, d). Contract (b, c, e).
		if (Case == 16) {
			a = x;
			d = y;

			for (int e = 0; e < N; ++e) {
				adj_value = adj[d * N + e];
				if (adj_value > 0) {
					b = e;
					c = e;
					sum += tensor[a * A + b * B + c * C + f] * adj_value;
				}
			}
		}	

		// Case 17 (46/50): (b, d). Contract (a, c, e).
		if (Case == 17) {
			b = x;
			d = y;

			for (int e = 0; e < N; ++e) {
				adj_value = adj[d * N + e];
				if (adj_value > 0) {
					a = e;
					c = e;
					sum += tensor[a * A + b * B + c * C + f] * adj_value;
				}
			}
		}

		// Case 18 (50/50): (d, e). Contract (a, b, c).
		if (Case == 18) {
			d = x;
			e = y;

			adj_value = adj[d * N + e];
			if (adj_value > 0) {
				for (int a = 0; a < N; ++a) {
					b = a;
					c = a;
					sum += tensor[a * A + b * B + c * C + f] * adj_value;
				}
			}
		}
		value[global_threadId] = sum;
	}
}


__global__ void RisiContraction_18_backward_job(float *tensor_gradient, float *adj, float *gradient, int N, int nChanels) {
	
	__shared__ int nContractions;
	__shared__ int X;
	__shared__ int Y;

	nContractions = NUM_CONTRACTIONS;

	int global_threadId = blockIdx.x * blockDim.x + threadIdx.x;

	if (global_threadId < N * N * N * nChanels) {
		X = N * nChanels * nContractions;
		Y = nChanels * nContractions;

		int f = global_threadId % nChanels;
		int c = (global_threadId / nChanels) % N;
		int b = ((global_threadId / nChanels) / N) % N;
		int a = ((global_threadId / nChanels) / N) / N;

		float sum = 0.0;

		int ind;
		float adj_value;

		for (int d = 0; d < N; ++d) {
			for (int e = 0; e < N; ++e) {
				adj_value = adj[d * N + e];

				if (adj_value > 0) {
					// +-----------+
					// | 1 + 1 + 1 |
					// +-----------+

					// Case 1 (1/50): Fix a, b. Contract c, d, e.
					ind = a * X + b * Y + 0 * nChanels + f;
					sum += gradient[ind] * adj_value;


					// Case 2 (3/50): Fix a, d. Contract b, c, e.
					ind = a * X + d * Y + 1 * nChanels + f;
					sum += gradient[ind] * adj_value;

					// Case 3 (5/50): Fix b, c. Contract a, d, e.
					ind = b * X + c * Y + 2 * nChanels + f;
					sum += gradient[ind] * adj_value;

					// Case 4 (6/50): Fix b, d. Contract a, c, e.
					ind = b * X + d * Y + 3 * nChanels + f;
					sum += gradient[ind] * adj_value;

					// Case 5 (10/50): Fix d, e. Contract a, b, c.
					ind = d * X + e * Y + 4 * nChanels + f;
					sum += gradient[ind] * adj_value;

					// +-------+
					// | 1 + 2 |
					// +-------+

					// Case 6 (11/50): (a, b). Contract (c, d). Singleton (e).
					if (c == d) {
						ind = a * X + b * Y + 5 * nChanels + f;
						sum += gradient[ind] * adj_value;
					}

					// Case 7 (13/50): (a, b). Contract (d, e). Singleton (c).
					if (d == e) {
						ind = a * X + b * Y + 6 * nChanels + f;
						sum += gradient[ind] * adj_value;
					}

					// Case 8 (17/50): (a, d). Contract (b, c). Singleton (e).
					if (b == c) {
						ind = a * X + d * Y + 7 * nChanels + f;
						sum += gradient[ind] * adj_value;
					}

					// Case 9 (18/50): (a, d). Contract (b, e). Singleton (c).
					if (b == e) {
						ind = a * X + d * Y + 8 * nChanels + f;
						sum += gradient[ind] * adj_value;
					}

					// Case 10 (23/50): (b, c). Contract (a, d). Singleton (e).
					if (a == d) {
						ind = b * X + c * Y + 9 * nChanels + f;
						sum += gradient[ind] * adj_value;
					}

					// Case 11 (26/50): (b, d). Contract (a, c). Singleton (e).
					if (a == c) {
						ind = b * X + d * Y + 10 * nChanels + f;
						sum += gradient[ind] * adj_value;
					}

					// Case 12 (27/50): (b, d). Contract (a, e). Singleton (c).
					if (a == e) {
						ind = b * X + d * Y + 11 * nChanels + f;
						sum += gradient[ind] * adj_value;
					}

					// Case 13 (28/50): (b, d). Contract (c, e). Singleton (a).
					if (c == e) {
						ind = b * X + d * Y + 12 * nChanels + f;
						sum += gradient[ind] * adj_value;
					}

					// Case 14 (38/50): (d, e). Contract (a, b). Singleton (c).
					if (a == b) {
						ind = d * X + e * Y + 13 * nChanels + f;
						sum += gradient[ind] * adj_value;
					}

					// Case 15 (40/50): (d, e). Contract (b, c). Singleton (a).
					if (b == c) {
						ind = d * X + e * Y + 14 * nChanels + f;
						sum += gradient[ind] * adj_value;
					}

					// +---+
					// | 3 |
					// +---+

					// Case 16 (43/50): (a, d). Contract (b, c, e).
					if ((b == c) && (c == e))  {
						ind = a * X + d * Y + 15 * nChanels + f;
						sum += gradient[ind] * adj_value;
					}

					// Case 17 (46/50): (b, d). Contract (a, c, e).
					if ((a == c) && (c == e))  {
						ind = b * X + d * Y + 16 * nChanels + f;
						sum += gradient[ind] * adj_value;
					}

					// Case 18 (50/50): (d, e). Contract (a, b, c).
					if ((a == b) && (b == c))  {
						ind = d * X + e * Y + 17 * nChanels + f;
						sum += gradient[ind] * adj_value;
					}
				}
			}
		}
		
		tensor_gradient[global_threadId] += sum;
	}
}


int rounded_division(int number1, int number2) {
	if (number1 % number2 == 0) {
		return number1 / number2;
	}
	return number1 / number2 + 1;
}

dim3 cuda_gridsize(int n){
	int k = (n - 1) / BLOCK + 1;
	int x = k;
	int y = 1;
	if (x > 65535){
		x = ceil(sqrt(k));
		y = (n - 1) / (x * BLOCK) + 1;
	}
	dim3 d(x, y, 1);
	return d;
}
		

void RisiContraction_18_forward_kernel(
	THCState* state,
	THCudaTensor* F_tensor,
	THCudaTensor* adj_tensor,
	THCudaTensor* output_tensor,
	int N,
	int nChannels
){
	
	float* F = THCudaTensor_data(state, F_tensor);
	float* adj = THCudaTensor_data(state, adj_tensor);
	float* output = THCudaTensor_data(state, output_tensor);
	hipStream_t stream = THCState_getCurrentStream(state);
	
	int size = N * N * nChannels * NUM_CONTRACTIONS;
	int nThreads = NUM_THREADS;
	dim3 dimGrid(rounded_division(size, nThreads));
	dim3 dimBlock(nThreads);
	hipError_t err;

	//RisiContraction_18_forward_job<<<dimGrid, dimBlock, 0, stream>>>(F, adj, output, N, nChannels);
	RisiContraction_18_forward_job<<<cuda_gridsize(size), BLOCK, 0, stream>>>(F, adj, output, N, nChannels);

	err = hipGetLastError();
	if (hipSuccess != err){
		fprintf(stderr, "CUDA kernel failed : %s\n", hipGetErrorString(err));
		exit(-1);
	}
}


void RisiContraction_18_backward_kernel(
	THCState* state,
	THCudaTensor* result_tensor,
	THCudaTensor* adj_tensor,
	THCudaTensor* gradient_tensor,
	int N,
	int nChannels
){
	
	float* gradient = THCudaTensor_data(state, gradient_tensor);
	float* adj = THCudaTensor_data(state, adj_tensor);
	float* result = THCudaTensor_data(state, result_tensor);
	hipStream_t stream = THCState_getCurrentStream(state);

	int size = N * N * nChannels;
	int nThreads = NUM_THREADS;
	dim3 dimGrid(rounded_division(size, nThreads));
	dim3 dimBlock(nThreads);
	hipError_t err;

	//RisiContraction_18_backward_job<<<dimGrid, dimBlock, 0, stream>>>(result, adj, gradient, N, nChannels);
	RisiContraction_18_backward_job<<<cuda_gridsize(size), BLOCK, 0, stream>>>(result, adj, gradient, N, nChannels);

	err = hipGetLastError();
	if (hipSuccess != err){
		fprintf(stderr, "CUDA kernel failed : %s\n", hipGetErrorString(err));
		exit(-1);
	}
}

#ifdef __cplusplus
	}
#endif
